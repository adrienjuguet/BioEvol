#include "hip/hip_runtime.h"
#include "Cuda.h"
#include <iostream>

/****************************CUDA****************************/
__global__
void kernel(GridCell *inGpuGrid, GridCell *outGpuGrid, long gridSize){
	/*for (int i = 0; i < width_*height_; i++) {
      grid_cell_[i]->diffuse_protein();
      grid_cell_[i]->degrade_protein();
  	}*/


}

void cuda_call4(int height, int width, GridCell** CpuGrid){
	
	long gridSize = sizeof(GridCell)*height*width;
	std::cout<<gridSize<<std::endl;
	GridCell * inGpuGrid;
	hipError_t ok = hipMalloc((void**) &inGpuGrid, gridSize);
	if(ok != hipSuccess)
	{
		std::cout << "Erreur d'allocation mémoire in !  Code d'erreur : "<< ok <<" : " << hipGetErrorString(ok)<< std::endl;
		return;
	}
	
	GridCell * outGpuGrid;
	ok = hipMalloc((void**) &outGpuGrid, gridSize);
	if(ok != hipSuccess)
	{
		std::cout << "Erreur d'allocation mémoire out !  Code d'erreur : "<< ok <<" : " << hipGetErrorString(ok)<< std::endl;
		return;
	}
	
	ok = hipMemcpy(inGpuGrid, CpuGrid, gridSize,hipMemcpyHostToDevice);
	if(ok != hipSuccess)
	{
		std::cout << "Erreur de copie mémoire in !  Code d'erreur : "<< ok <<" : " << hipGetErrorString(ok)<< std::endl;
		return;
	}
	
	dim3 dimBlock(32,32);
	dim3 dimGrid(width/dimBlock.x, height/dimBlock.y);
	
	kernel<<<dimGrid, dimBlock>>>(inGpuGrid, outGpuGrid, gridSize);
	
	hipDeviceSynchronize();
	
	ok = hipGetLastError();
	if(ok != hipSuccess)
	{
		std::cout << "Erreur du kernel !  Code d'erreur : "<< ok <<" : " << hipGetErrorString(ok)<< std::endl;
		return;
	}
		
	ok = hipMemcpy(CpuGrid, outGpuGrid, gridSize, hipMemcpyDeviceToHost);
	if(ok != hipSuccess)
	{
		std::cout << "Erreur de copie mémoire out !  Code d'erreur : "< <ok <<" : " << hipGetErrorString(ok)<< std::endl;
		return;
	}
	
	ok = hipFree(inGpuGrid);
	if(ok != hipSuccess)
	{
		std::cout << "Erreur de libération mémoire in !  Code d'erreur : "<< ok <<" : " << hipGetErrorString(ok) << std::endl;
		return;
	}
	ok = hipFree(outGpuGrid);
	if(ok != hipSuccess)
	{
		std::cout << "Erreur de libération mémoire out !  Code d'erreur : "<< ok <<" : " << hipGetErrorString(ok)<< std::endl;
		return;
	}

}


/************************************************************/
