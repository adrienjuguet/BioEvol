#include "hip/hip_runtime.h"
#include "Cuda.h"
#include <iostream>

/****************************CUDA****************************/
__global__
void kernel_sum_metabolic_error(GridCell * inGpuGridCell, float * inGpuMetabolic,float * outSum, int metabolic_size)
{
	/*
  for (int i = 0; i < Common::Metabolic_Error_Precision; i++) {
    sum_metabolic_error+=std::abs(gridcell_->environment_target[i]-metabolic_error[i]);
  }
  	*/
}

void cuda_call_sum_metabolic_error(int metabolic_size, GridCell* gridcell, float* metabolic)
{		
	
	long metabolicSize = sizeof(float)*metabolic_size;
	
	GridCell* inGpuGridCell;
	float * inGpuMetabolic;
	float * outSum;
	hipError_t ok;
	
	/**Allocation mémoire GPU**/
	ok = hipMalloc((void**) &inGpuGridCell, sizeof(GridCell*));
	if(ok != hipSuccess)
	{
		std::cout << "Erreur d'allocation mémoire inGpuGridCell !  Code d'erreur : "<< ok <<" : " << hipGetErrorString(ok)<< std::endl;
		return;
	}
	
	ok = hipMalloc((void**) &inGpuMetabolic, metabolicSize);
	if(ok != hipSuccess)
	{
		std::cout << "Erreur d'allocation mémoire inGpuMetabolic !  Code d'erreur : "<< ok <<" : " << hipGetErrorString(ok)<< std::endl;
		return;
	}

	ok = hipMalloc((void**) &outSum, sizeof(float));
	if(ok != hipSuccess)
	{
		std::cout << "Erreur d'allocation mémoire out !  Code d'erreur : "<< ok <<" : " << hipGetErrorString(ok)<< std::endl;
		return;
	}
	/**************************/
	
	/**Init inGpuGridCell & inGpuMetabolic**/
	ok = hipMemcpy(inGpuGridCell, gridcell, sizeof(GridCell*),hipMemcpyHostToDevice);
	if(ok != hipSuccess)
	{
		std::cout << "Erreur de copie mémoire inGpuGridCell !  Code d'erreur : "<< ok <<" : " << hipGetErrorString(ok)<< std::endl;
		return;
	}
	ok = hipMemcpy(inGpuMetabolic, metabolic, metabolicSize,hipMemcpyHostToDevice);
	if(ok != hipSuccess)
	{
		std::cout << "Erreur de copie mémoire inGpuMetabolic !  Code d'erreur : "<< ok <<" : " << hipGetErrorString(ok)<< std::endl;
		return;
	}
	/******************/

	dim3 dimBlock(32);
	dim3 dimGrid(metabolicSize/dimBlock.x);
	
	kernel_sum_metabolic_error<<<dimGrid, dimBlock>>>(inGpuGridCell, inGpuMetabolic, outSum, metabolic_size);
	
	hipDeviceSynchronize();
	ok = hipGetLastError();
	if(ok != hipSuccess)
	{
		std::cout << "Erreur du kernel !  Code d'erreur : "<< ok <<" : " << hipGetErrorString(ok)<< std::endl;
		return;
	}
		
	/**Récupération des valeurs sur le CPU**/
	/*double * sum;
	ok = hipMemcpy(sum, outSum, sizeof(float), hipMemcpyDeviceToHost);
	if(ok != hipSuccess)
	{
		std::cout << "Erreur de copie mémoire out !  Code d'erreur : "<< ok <<" : " << hipGetErrorString(ok)<< std::endl;
		return;
	}*/
	/***************************************/
	
	/**Libération de la mémoire**/
	ok = hipFree(inGpuGridCell);
	if(ok != hipSuccess)
	{
		std::cout << "Erreur de libération mémoire inGpuGridCell !  Code d'erreur : "<< ok <<" : " << hipGetErrorString(ok) << std::endl;
		return;
	}
	ok = hipFree(inGpuMetabolic);
	if(ok != hipSuccess)
	{
		std::cout << "Erreur de libération mémoire inGpuMetabolic !  Code d'erreur : "<< ok <<" : " << hipGetErrorString(ok)<< std::endl;
		return;
	}
	ok = hipFree(outSum);
	if(ok != hipSuccess)
	{
		std::cout << "Erreur de libération mémoire outSum !  Code d'erreur : "<< ok <<" : " << hipGetErrorString(ok)<< std::endl;
		return;
	}
	/****************************/
}


/************************************************************/
