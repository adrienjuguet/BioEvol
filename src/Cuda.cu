#include "hip/hip_runtime.h"
#include "Cuda.h"
#include <iostream>

/****************************CUDA****************************/
__global__
void kernel(GridCell *inGpuGrid, GridCell *outGpuGrid, long gridSize){
	/*for (int i = 0; i < width_*height_; i++) {
      grid_cell_[i]->diffuse_protein();
      grid_cell_[i]->degrade_protein();
  	}*/


}

void cuda_call4(int height, int width, GridCell** CpuGrid)
{	
	long gridSize = sizeof(GridCell)*height*width;
        GridCell * inGpuGrid;
        GridCell * outGpuGrid;
        hipError_t ok;
	
	/**Allocation mémoire GPU**/
	ok = hipMalloc((void**) &inGpuGrid, gridSize);
	if(ok != hipSuccess)
	{
		std::cout << "Erreur d'allocation mémoire in !  Code d'erreur : "<< ok <<" : " << hipGetErrorString(ok)<< std::endl;
		return;
	}

	ok = hipMalloc((void**) &outGpuGrid, gridSize);
	if(ok != hipSuccess)
	{
		std::cout << "Erreur d'allocation mémoire out !  Code d'erreur : "<< ok <<" : " << hipGetErrorString(ok)<< std::endl;
		return;
	}
	/**************************/
	
	/**Init inGpuGrid**/
	ok = hipMemcpy(inGpuGrid, CpuGrid, gridSize,hipMemcpyHostToDevice);
	if(ok != hipSuccess)
	{
		std::cout << "Erreur de copie mémoire in !  Code d'erreur : "<< ok <<" : " << hipGetErrorString(ok)<< std::endl;
		return;
	}
	/******************/

	dim3 dimBlock(32,32);
	dim3 dimGrid(width/dimBlock.x, height/dimBlock.y);
	
	kernel<<<dimGrid, dimBlock>>>(inGpuGrid, outGpuGrid, gridSize);
	
	hipDeviceSynchronize();
	ok = hipGetLastError();
	if(ok != hipSuccess)
	{
		std::cout << "Erreur du kernel !  Code d'erreur : "<< ok <<" : " << hipGetErrorString(ok)<< std::endl;
		return;
	}
		
	/**Récupération des valeurs sur le CPU**/
	/*ok = hipMemcpy(CpuGrid, outGpuGrid, gridSize, hipMemcpyDeviceToHost);
	if(ok != hipSuccess)
	{
		std::cout << "Erreur de copie mémoire out !  Code d'erreur : "<<ok <<" : " << hipGetErrorString(ok)<< std::endl;
		return;
	}*/
	/***************************************/
	
	/**Libération de la mémoire**/
	ok = hipFree(inGpuGrid);
	if(ok != hipSuccess)
	{
		std::cout << "Erreur de libération mémoire in !  Code d'erreur : "<< ok <<" : " << hipGetErrorString(ok) << std::endl;
		return;
	}
	ok = hipFree(outGpuGrid);
	if(ok != hipSuccess)
	{
		std::cout << "Erreur de libération mémoire out !  Code d'erreur : "<< ok <<" : " << hipGetErrorString(ok)<< std::endl;
		return;
	}
	/****************************/
}


/************************************************************/
