#include "hip/hip_runtime.h"
#include "Cuda.h"
#include <iostream>

/****************************CUDA****************************/
__global__
void kernel_sum_metabolic_error(GridCell * inGpuGridCell, float * inGpuMetabolic,float * outSum, int * metabolic_size)
{
  	uint threadId = blockIdx.x * blockDim.x + threadIdx.x;
  	
  	float sum = 0;
  	if(threadId < *metabolic_size){
		sum = inGpuGridCell->environment_target[threadId]-inGpuMetabolic[threadId];
		if(sum < 0){
			sum *= -1;
		}
	}
	atomicAdd(outSum,sum);
}

float cuda_call_sum_metabolic_error(int metabolic_size, GridCell* gridcell, float* metabolic)
{		
	
	long metabolicSize = sizeof(float)*metabolic_size;
	
	GridCell* inGpuGridCell;
	float * inGpuMetabolic;
	int * inGpuMetabolicSize;
	float * outSum;
	hipError_t ok;
	
	/**Allocation mémoire GPU**/
	ok = hipMalloc((void**) &inGpuGridCell, sizeof(GridCell*));
	if(ok != hipSuccess)
	{
		std::cout << "Erreur d'allocation mémoire inGpuGridCell !  Code d'erreur : "<< ok <<" : " << hipGetErrorString(ok)<< std::endl;
		return -1;
	}
	
	ok = hipMalloc((void**) &inGpuMetabolic, metabolicSize);
	if(ok != hipSuccess)
	{
		std::cout << "Erreur d'allocation mémoire inGpuMetabolic !  Code d'erreur : "<< ok <<" : " << hipGetErrorString(ok)<< std::endl;
		return -1;
	}

	ok = hipMalloc((void**) &outSum, sizeof(float*));
	if(ok != hipSuccess)
	{
		std::cout << "Erreur d'allocation mémoire out !  Code d'erreur : "<< ok <<" : " << hipGetErrorString(ok)<< std::endl;
		return -1;
	}
	ok = hipMalloc((void**) &inGpuMetabolicSize, sizeof(int));
        if(ok != hipSuccess)
        {
                std::cout << "Erreur d'allocation mémoire metabolic_size !  Code d'erreur : "<< ok <<" : " << hipGetErrorString(ok)<< std::endl;
                return -1;
        }

	/**************************/
	
	/**Init inGpuGridCell & inGpuMetabolic**/
	ok = hipMemcpy(inGpuGridCell, gridcell, sizeof(GridCell*),hipMemcpyHostToDevice);
	if(ok != hipSuccess)
	{
		std::cout << "Erreur de copie mémoire inGpuGridCell !  Code d'erreur : "<< ok <<" : " << hipGetErrorString(ok)<< std::endl;
		return -1;
	}
	ok = hipMemcpy(inGpuMetabolic, metabolic, metabolicSize,hipMemcpyHostToDevice);
	if(ok != hipSuccess)
	{
		std::cout << "Erreur de copie mémoire inGpuMetabolic !  Code d'erreur : "<< ok <<" : " << hipGetErrorString(ok)<< std::endl;
		return -1;
	}
	int * ptr_metabolic_size = &metabolic_size; 
	ok = hipMemcpy(inGpuMetabolicSize, ptr_metabolic_size, sizeof(int),hipMemcpyHostToDevice);
        if(ok != hipSuccess)
        {
                std::cout << "Erreur de copie mémoire metabolic_size !  Code d'erreur : "<< ok <<" : " << hipGetErrorString(ok)<< std::endl;
                return -1;
        }

	/******************/

	dim3 dimBlock(32);
	dim3 dimGrid(metabolicSize/dimBlock.x);
	
	std::cout << "starting kernel :" << ok << std::endl;	
	kernel_sum_metabolic_error<<<dimGrid, dimBlock>>>(inGpuGridCell, inGpuMetabolic, outSum, inGpuMetabolicSize);
	std::cout << "end kernel :" << ok << std::endl;
	
	getchar();
	hipDeviceSynchronize();
	std::cout << "end thread synchronize :" << ok << std::endl;

	ok = hipGetLastError();
	if(ok != hipSuccess)
	{
		std::cout << "Erreur du kernel !  Code d'erreur : "<< ok <<" : " << hipGetErrorString(ok)<< std::endl;
		return -1;
	}
		
	/**Récupération des valeurs sur le CPU**/
	float * sum = 0;
	ok = hipMemcpy(sum, outSum, sizeof(float*), hipMemcpyDeviceToHost);
	if(ok != hipSuccess)
	{
		std::cout << "Erreur de copie mémoire out !  Code d'erreur : "<< ok <<" : " << hipGetErrorString(ok)<< std::endl;
		std::cout << "sum :" << *sum << std::endl;
		std::cout << "outSum :" << *outSum << std::endl;		
		return -1;
	}
	/***************************************/
	
	/**Libération de la mémoire**/
	ok = hipFree(inGpuGridCell);
	if(ok != hipSuccess)
	{
		std::cout << "Erreur de libération mémoire inGpuGridCell !  Code d'erreur : "<< ok <<" : " << hipGetErrorString(ok) << std::endl;
		return -1;
	}
	ok = hipFree(inGpuMetabolic);
	if(ok != hipSuccess)
	{
		std::cout << "Erreur de libération mémoire inGpuMetabolic !  Code d'erreur : "<< ok <<" : " << hipGetErrorString(ok)<< std::endl;
		return -1;
	}
	ok = hipFree(outSum);
	if(ok != hipSuccess)
	{
		std::cout << "Erreur de libération mémoire outSum !  Code d'erreur : "<< ok <<" : " << hipGetErrorString(ok)<< std::endl;
		return -1;
	}
	ok = hipFree(inGpuMetabolicSize);
        if(ok != hipSuccess)
        {
                std::cout << "Erreur de libération mémoire metabolic_size !  Code d'erreur : "<< ok <<" : " << hipGetErrorString(ok) << std::endl;
                return -1;
        }

	/****************************/
	return * sum;
}


/************************************************************/
