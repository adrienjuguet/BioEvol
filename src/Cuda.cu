#include "hip/hip_runtime.h"
#include "Cuda.h"
#include <iostream>

/****************************CUDA****************************/
__global__
void kernel_organism_dyingornot(Protein **inGpuProtein, double outGpuProtein, long proteinSize){
	/*for (int i = 0; i < width_*height_; i++) {
      grid_cell_[i]->diffuse_protein();
      grid_cell_[i]->degrade_protein();
  	}*/
}

void cuda_call_protein(std::vector<Protein*> cpuProtein)
{	
	Protein** protein_iterator = &cpuProtein[0];
	long vectorSize = cpuProtein.size();
	
	long proteinSize = sizeof(Protein*)*vectorSize;
        Protein ** inGpuProtein;
        double * outGpuProtein;
        hipError_t ok;
	
	/**Allocation mémoire GPU**/
	ok = hipMalloc((void**) &inGpuProtein, proteinSize);
	if(ok != hipSuccess)
	{
		std::cout << "Erreur d'allocation mémoire in !  Code d'erreur : "<< ok <<" : " << hipGetErrorString(ok)<< std::endl;
		return;
	}

	ok = hipMalloc((void**) &outGpuProtein, sizeof(double));
	if(ok != hipSuccess)
	{
		std::cout << "Erreur d'allocation mémoire out !  Code d'erreur : "<< ok <<" : " << hipGetErrorString(ok)<< std::endl;
		return;
	}
	/**************************/
	
	/**Init inGpuProtein**/
	ok = hipMemcpy(inGpuProtein, protein_iterator, proteinSize,hipMemcpyHostToDevice);
	if(ok != hipSuccess)
	{
		std::cout << "Erreur de copie mémoire in !  Code d'erreur : "<< ok <<" : " << hipGetErrorString(ok)<< std::endl;
		return;
	}
	/******************/

	dim3 dimBlock(32);
	dim3 dimGrid(proteinSize/dimBlock.x);
	
	kernel_organism_dyingornot<<<dimGrid, dimBlock>>>(inGpuProtein, * outGpuProtein, proteinSize);
	
	hipDeviceSynchronize();
	ok = hipGetLastError();
	if(ok != hipSuccess)
	{
		std::cout << "Erreur du kernel !  Code d'erreur : "<< ok <<" : " << hipGetErrorString(ok)<< std::endl;
		return;
	}
		
	/**Récupération des valeurs sur le CPU**/
	double * sum;
	ok = hipMemcpy(sum, outGpuProtein, proteinSize, hipMemcpyDeviceToHost);
	if(ok != hipSuccess)
	{
		std::cout << "Erreur de copie mémoire out !  Code d'erreur : "<< ok <<" : " << hipGetErrorString(ok)<< std::endl;
		return;
	}
	/***************************************/
	
	/**Libération de la mémoire**/
	ok = hipFree(inGpuProtein);
	if(ok != hipSuccess)
	{
		std::cout << "Erreur de libération mémoire in !  Code d'erreur : "<< ok <<" : " << hipGetErrorString(ok) << std::endl;
		return;
	}
	ok = hipFree(outGpuProtein);
	if(ok != hipSuccess)
	{
		std::cout << "Erreur de libération mémoire out !  Code d'erreur : "<< ok <<" : " << hipGetErrorString(ok)<< std::endl;
		return;
	}
	/****************************/
}


/************************************************************/
